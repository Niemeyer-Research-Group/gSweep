#include "hip/hip_runtime.h"
/** 
    Copyright (C) 2017 Kyle Niemeyer, niemeyek@oregonstate.edu AND
    Daniel Magee, mageed@oregonstate.edu
*/
/*
    This file is distribued under the MIT License.  See LICENSE at top level of directory or: <https://opensource.org/licenses/MIT>.
*/

// Fuckin duh.  You can't do it like this you need an array!

#include "waveConsts.h"

__device__  
void stepUpdate(states *state, int idx[3], int tstep)
{
    int ins = tstep&1; // In step with
    stencil(state, idx, ins);
}


__device__ void boundary(int gid, int tid, int *idxes)
{
    idxes[1] = tid;
    if (deqConsts.typ == 0)
    {
        idxes[0] = (gid) ? tid : deqconsts.idxF;
        idxes[2] = (gid == deqConsts.idxF) ? 0 : tid;   
    }
    // // Reflective
    // else
    // {
    //     if (gid == 1) state[0] = state[2];
    //     if (gid = deqConsts.idxF-1) state[deqConsts.idxF] = state[deqConsts.idxF-2]; 
    // }
}

__global__ 
void classicStep(states *state, int nX, int tstart, int tend)
{
    grid_group grid = this_grid();
    int idxes[3];

    for (int k = tstart; k<tend; k++)
    {
        for(int i = grid.thread_rank(); i < nX; i += grid.size())
        {
            boundary(i, i, &idxes);
            stepUpdate(state, idxes, k);
        }
    }
    grid.sync()
}

__device__
__forceinline__
void sweepRead(states *tState, states *statein, int gid, int tid, int bd)
{
    int tadj = tid * (bd + 1);
    tState[tid+1] = statein[gid];
    __syncthreads();
    if (tid<2)
    {
        if (gid == 0)
        {
            tState[0] = statesin[deqConsts.idxF];
        }   
        else if (gid == deqConsts.idxF)
        {
            tState[blockDim.x + 1] = statesin[0];
        }
        else
        {
            tState[tadj] = statesin[(gid-1) + tadj];
        }
    }
}

__global__
void upTriangle(states *statein, states *stateout, int tstep)
{   
    extern __shared__ states tState[];

    //Global Thread ID
    int gid = blockDim.x * blockIdx.x + threadIdx.x; 
    int tidx = threadIdx.x; //Block Thread ID
    int mid = blockDim.x >> 1;
    int gidout = (gid - mid) % deqConsts.idxF;
    int tnow = tstep;
    int idxes[3];
    for (int k=-1; k<2; k++) idxes[k+1] = tid + k;

    tState[tidx] = statesin[gid];

    __syncthreads();

    for (int k=1; k<mid; k++)
    {
        if (tidx < (blockDim.x-k) && tidx >= k)
        {
            stepUpdate(tState, idxes, tnow); 
        }
        tnow++;
        __syncthreads();
    }
    stateout[gidout] = tState[tidx];
}

__global__
void downTriangle(states *statein, states *stateout, int tstep)
{
    extern __shared__ states tState[];

    int tid = threadIdx.x; // Thread index
    int mid = blockDim.x >> 1; // Half of block size
    int base = blockDim.x + 2; 
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int tidx = tid + 1;
    int tnow = tstep; // read tstep into register.
    int idxes[3];
    for (int k=-1; k<2; k++) idxes[k+1] = tidx + k;

    sweepRead(tState, statein, gid, tid, blockDim.x);
    
    for (int k=mid; k>0; k--)
    {
        if (tidx < (base-k) && tidx >= k)
        {
            stepUpdate(temper, idxes, tnow);
        }
        tnow++;
        __syncthreads();
    }
    state[gid] = temper[tidx];
}

__global__
void wholeTriangle(states *statein, states *stateout, int tstep, int dir)
{
    extern __shared__ states tState[];

    int tid = threadIdx.x; // Thread index
    int mid = blockDim.x >> 1; // Half of block size
    int base = blockDim.x + 2; 
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int gidout = (gid + dir*mid) % deqConsts.idxF;
    int tidx = tid + 1;
    int tnow = tstep; // read tstep into register.
    int idxes[3];
    for (int k=-1; k<2; k++) idxes[k+1] = tidx + k;

    sweepRead(tState, statein, gid, tid, blockDim.x);

    for (int k=mid; k>0; k--)
	{
		if (tidx < (base-k) && tidx >= k)
		{
        	stepUpdate(temper, tidx, tnow);
		}
		tnow++;
		__syncthreads();
	}

	for (int k=2; k<=mid; k++)
	{
		if (tidx < (base-k) && tidx >= k)
		{
            stepUpdate(temper, tidx, tnow);
		}
		tnow++;
		__syncthreads();
    }
    state[gidout] = temper[tidx];
}

void classicWrapper(states *state, int *tstep)
{
    cout << "Classic scheme" << endl;
    states *stin;
    int tBytes = cGlob.szState*cGlob.nX;
    hipMalloc((void **)&stin, tBytes);

    // Copy the initial conditions to the device array.
    hipMemcpy(dstin, state, tBytes, hipMemcpyHostToDevice);

    double t_eq = NSTEPS * cGlob.dt;
    double twrite = cGlob.freq - 0.25*cGlob.dt;

    classicStep <<< cGlob.bks, cGlob.tpb >>> (stin, tmine);
    t_eq += cGlob.dt;
    tmine++;

    hipMemcpy(state, stin, tBytes, hipMemcpyDeviceToHost);
    hipFree(stin);
    return t_eq;
}

void sweptWrapper(states *state, int *tstep)
{
    cout << "Swept scheme" << endl;
    REAL *stateA, *stateB;
    int tmine = *tstep;
    const int tBytes = cGlob.szState*cGlob.nX;
    const size_t smem = (cGlob.tpb + 2) * cGlob.szState;

    hipMalloc((void **)&stateA, tBytes);
    hipMalloc((void **)&stateB, tBytes);

    // Copy the initial conditions to the device array.
    hipMemcpy(stateA, state, tBytes, hipMemcpyHostToDevice);

    double t_eq = NSTEPS * cGlob.dt;
    double twrite = cGlob.freq - 0.25*cGlob.dt;
    //inline dir = -1, split dir = 1 because passing after calculation.
    upTriangle <<< cGlob.bks, cGlob.tpb, smem >>> (stateA, stateB, tmine);
    wholeDiamond <<< cGlob.bks, cGlob.tpb, smem >>> (stateB, stateA, tmine, 1);

    while (t_eq <= cGlob.tf)
    {
        wholeDiamond <<< cGlob.bks, cGlob.tpb, smem >>> (stateA, stateB, tmine, -1);
        t_eq += cGlob.dt;
        tmine++;

        if (t_eq > twrite)
        {
            downTriangle <<< cGlob.bks, cGlob.tpb, smem >>> (stateB, stateA, tmine, 1);
            hipMemcpy(state, stateA, tBytes, hipMemcpyDeviceToHost);
            twrite += cGlob.freq;
        }
    }
    hipMemcpy(state, dks_in, tBytes, hipMemcpyDeviceToHost);
    hipFree(dks_in);
    return t_eq;
}
